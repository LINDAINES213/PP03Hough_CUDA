#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 nvcc houghBase.cu -o hough
 ./hough runway.pgm
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "common/pgm.h"
#include "common/PGMImage.h"
#include <cairo/cairo.h>

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// TODO usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
//__constant__ float d_Cos[degreeBins];
//__constant__ float d_Sin[degreeBins];

//*****************************************************************
//TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }
//TODO Kernel memoria Constante
// __global__ void GPU_HoughTranConst(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  //TODO calcular: int gloID = ?
  //int gloID = w * h + 1; //TODO
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados

}

void drawHoughLinesCairo(cairo_t *cr, int* hough_acc, float rMax, float rScale, int degreeBins, int rBins, float radInc, int threshold, int w, int h) {
    for (int rIdx = 0; rIdx < rBins; rIdx++) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            int acc_val = hough_acc[rIdx * degreeBins + tIdx];
            if (acc_val > threshold) { // Solo dibujar líneas con alta acumulación
                float theta = tIdx * radInc;
                float r = rIdx * rScale - rMax;

                // Convertir (r, theta) en puntos (x1, y1) y (x2, y2) para dibujar la línea
                float cosTheta = cos(theta);
                float sinTheta = sin(theta);
                int x0 = r * cosTheta;
                int y0 = r * sinTheta;

                // Puntos extremos de la línea para dibujar en la imagen
                int x1 = std::round(x0 + 1000 * (-sinTheta));
                int y1 = std::round(y0 + 1000 * (cosTheta));
                int x2 = std::round(x0 - 1000 * (-sinTheta));
                int y2 = std::round(y0 - 1000 * (cosTheta));

                // Dibujar la línea en la imagen con Cairo
                cairo_move_to(cr, x1, y1);
                cairo_line_to(cr, x2, y2);
                cairo_set_source_rgb(cr, 1.0, 0.0, 0.0); // Color rojo
                cairo_set_line_width(cr, 1);
                cairo_stroke(cr);  // Dibujar la línea
            }
        }
    }
}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  float* d_Cos;
  float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global
  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Tiempo de ejecución del kernel: %f ms\n", elapsedTime);

   // Crear una superficie de imagen Cairo
  cairo_surface_t *surface = cairo_image_surface_create(CAIRO_FORMAT_RGB24, w, h);
  cairo_t *cr = cairo_create(surface);

  // Dibujar las líneas detectadas usando Cairo
  int threshold = 1500;
  drawHoughLinesCairo(cr, h_hough, rMax, rScale, degreeBins, rBins, radInc, threshold, w, h);

  // Guardar la imagen resultante en formato PNG
  cairo_surface_write_to_png(surface, "imagen_con_lineas.png");

  // Limpiar recursos
  cairo_destroy(cr);
  cairo_surface_destroy(surface);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // TODO clean-up
  hipFree(d_Cos);
  hipFree(d_Sin);
  hipFree(d_in);
  hipFree(d_hough);

  free(pcCos);
  free(pcSin);
  free(h_hough);
  free(cpuht);

  return 0;
}
